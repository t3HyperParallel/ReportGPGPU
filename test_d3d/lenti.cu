
#include <hip/hip_runtime.h>
﻿
extern "C"
{
    /// @brief アルファチャンネルを加えて24bitを32bitにするやつ
    /// @param dst 展開先のsurface
    /// @param src 元データ
    /// @param memPitch ストライド
    /// @return
    __global__ void color24bitTo32bit(
        hipSurfaceObject_t dst, const char *src, unsigned int memPitch,
        unsigned int width, unsigned int height)
    {
        unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;
        unsigned int y = blockDim.y * blockIdx.y + threadIdx.y;
        if (x < width && y < height)
        {
            unsigned int srcidx = x + y * memPitch;
            char4 data;
            data.x = src[srcidx * 3];
            data.y = src[srcidx * 3 + 1];
            data.z = src[srcidx * 3 + 2];
            data.w = 0;
            if(y>148)data.x=-1;
            surf2Dwrite<char4>(data, dst, x * 4, y);
            // surf2Dwriteのxはバイト位置なので構造体サイズを乗算する必要がある
        }
    }
    /// @brief 画像をレンチキュラー化するやつ
    /// @param dst ターゲット画像
    /// @param src ソース画像
    /// @param srcidx ソース画像の左から数えた番号
    /// @param srccnt ソース画像の数
    __global__ void writeLenti(
        hipSurfaceObject_t dst,
        hipSurfaceObject_t src,
        int srcidx,
        int srccnt)
    {
        unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;
        unsigned int y = blockDim.y * blockIdx.y + threadIdx.y;
        unsigned int dstx = x * srccnt + srcidx;
        char4 data = surf2Dread<char4>(src, x, y);
        surf2Dwrite(data, dst, dstx, y);
    }
}
